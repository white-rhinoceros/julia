#include "hip/hip_runtime.h"
#include "dot_gpu.h"

__global__ void dot__(float *v1, float *v2, float *res, int N) {
    __shared__ float cache [threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0.0;
    while (tid < N) {
        temp += v1[tid] * v2[tid];
        tid += blockDim.x * gridDim.x;
    }
    cache[cacheIndex] = temp;

    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        res[blockIdx.x] = cache[0];
    }
}

float * gpu_dot (float *v1, float *v2, size_t N) {
    float *dev_v1, *dev_v2, *dev_res, *res;

    res = new float[blocksPerGrid];

    hipMalloc((void**)&dev_v1, N * sizeof(float));
    hipMalloc((void**)&dev_v2, N * sizeof(float));
    hipMalloc((void**)&dev_res, blocksPerGrid * sizeof(float));

    hipMemcpy(dev_v1, v1, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_v2, v2, N * sizeof(float), hipMemcpyHostToDevice);

    dot__<<<blocksPerGrid, threadsPerBlock>>>(dev_v1, dev_v2, dev_res, (int)N);
    hipMemcpy(res, dev_res, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_v1);
    hipFree(dev_v2);
    hipFree(dev_res);

    return res;
}